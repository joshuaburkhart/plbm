#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "./lib/lapack.h"
#include "./lib/arraylib.h"

__global__ void cuda_kron(double *out,int min_m,int min_n,double *A,int ma,int na,double *B,int mb,int nb)
{

  int i = blockIdx.x;
  int j = threadIdx.x;
  out[i*na*nb +j]=*(A+(i/min_m*na+j/min_n)) * *(B+(i%min_m*nb+j%min_n));

}

void matrx_inv(double out[],double *A,int n) {

    tran(out,A,n,n);

    ptrdiff_t N=n;
    ptrdiff_t M=n;
    ptrdiff_t lda=n;
    ptrdiff_t ipiv[N];
    ptrdiff_t info=0;
    ptrdiff_t lwork=n*n;
    double work[lwork];

    dgetrf(&M,&N,out,&lda,ipiv,&info);
    if(info!=0) {
        //printf("dgetrf returns info code %i ... inverse could not be calculated\n",info);
        //printf("M:   %i\n",M);
        //printf("N:   %i\n",N);
        //printf("lda: %i\n",lda);
        info=0;
    }

    dgetri(&N,out,&lda,ipiv,work,&lwork,&info);
    if(info!=0) {
        //printf("dgetri returns info code %i ... inverse could not be calculated\n",info);
        //printf("N:   %i\n",N);
        //printf("lda: %i\n",lda);
    }

    tran(out,out,n,n);
    return;
}

void array_rdv(double out[],double *A,int m,int n,double d) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j] =*(A+(i*n+j)) / d;
        }
    }
    return;
}

void matrx_sub3(double out[],double *A,int m,int n,double d) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j] =*(A+(i*n+j)) - d;
        }
    }
    return;
}

void matrx_sub(double out[],double d,double *A,int m,int n) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j] =d - *(A+(i*n+j));
        }
    }
    return;
}

void matrx_mlt2(double out[],double *A,int ma,int na,double *B,int mb,int nb) {

    #pragma omp parallel for
    for(int i=0; i<ma; i++) {
        for(int j=0; j<nb; j++) {
            double sum=0;
            for(int k=0; k<na; k++) {
                sum+=*(A+(i*na+k)) * *(B+(j+k*nb));
            }
            out[i*nb+j] =sum;
        }
    }
    return;
}

void matrx_mlt(double out[],double d,double *A,int m,int n) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j]  = *(A+(i*n+j)) * d;
        }
    }
    return;
}

void array_mlt(double out[],double A[],int m,int n,double *B) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j] =A[i*n+j] * *(B+(i*n+j));
        }
    }
    return;
}

void array_pow(double out[],double d,double *A,int m,int n) {

    #pragma omp parallel for
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i*n+j] =pow(d,*(A+(i*n+j)));
        }
    }
    return;
}

double matrx_det(double *A,int n) {
    double luout[n*n];
    tran(luout,A,n,n);
    ptrdiff_t N=n;
    ptrdiff_t M=n;
    ptrdiff_t lda=n;
    ptrdiff_t ipiv[N];
    ptrdiff_t info=0;
    //ptrdiff_t lwork=N*N;
    //double work[lwork];
    dgetrf(&M,&N,luout,&lda,ipiv,&info);
    if(info!=0) {
        //printf("dgetrf returns info code %i ... determinant cannot be calculated\n",info);
        //printf("M:   %i\n",M);
        //printf("N:   %i\n",N);
        //printf("lda: %i\n",lda);
    }
    double luout2[n*n];
    tran(luout2,luout,n,n);
    double diag=1;
    for(int i = 0; i < n; i++) {
        diag *= luout2[i * n + i];
    }
    #pragma omp parallel for
    for(int i = 0; i < n; i++) {
        luout2[i *  n + i]  = 1;
        for(int j = i+1; j < n; j++) {
            luout2[i *  n + j]  = 0;
        }
    }
    double dtm=det_l(luout2,n);
    return(dtm * diag);
}

static double det_l(double *A,int n) {
    int i, j, k;
    double m[n][n];
    double det = 1;
    #pragma omp parallel for
    for (int x = 0; x < n; x++ ) {
        for (int y = 0; y < n; y++ ) {
            m[x][y] = *(A+(x*n+y));
        }
    }
    for ( k = 0; k < n; k++ ) {
        if ( m[k][k] == 0 ) {
            int ok = 0;
            for ( j = k; j < n; j++ ) {
                if (m[j][k] != 0 ) {
                    ok = 1;
                }
            }
            if (ok==0) {
                return 0;
            }
            for ( i = k; i < n; i++ ) {
                double tmp= m[i][j];
                m[i][j]=m[i][k];
                m[i][k]=tmp;
            }
            det = -det;
        }
        det *= m[k][k];
        if ( k + 1 < n ) {
            for ( i = k + 1; i < n; i++ ) {
                for ( j = k + 1; j < n; j++ ) {
                    m[i][j] = m[i][j] - m[i][k] * m[k][j] / m[k][k];
                }
            }
        }
    }
    return det;
}

void ones(double out[],int m,int n) {

    #pragma omp parallel for
    for(int i=0; i<m*n; i++) {
        out[i]=1;
    }
    return;
}

void kron(double *out,double *A,int ma,int na,double *B,int mb,int nb) {

    int min_m = ma < mb ? ma : mb;
    int min_n = na < nb ? na : nb;
    /*
    for(int i=0; i<ma*mb; i++) {
        for(int j=0; j<na*nb; j++) {
            out[i*na*nb +j]=*(A+(i/min_m*na+j/min_n)) * *(B+(i%min_m*nb+j%min_n));
        }
    }
    */
    size_t A_size = ma*na*sizeof(double);
    size_t B_size = mb*nb*sizeof(double);
    size_t out_size = ma*mb*na*nb*sizeof(double);
    double *d_A,*d_B,*d_out;
    hipMalloc(&d_A,A_size);
    hipMalloc(&d_B,B_size);
    hipMalloc(&d_out,out_size);

    hipMemcpy(d_A,A,A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,B_size, hipMemcpyHostToDevice);

    int threadsPerBlock = na*nb;
    int blocksPerGrid = ma*mb;

    cuda_kron<<<blocksPerGrid, threadsPerBlock>>>(d_out,min_m,min_n,d_A,ma,na,d_B,mb,nb);

    hipMemcpy(out,d_out,out_size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);

    return;
}

void tran(double out[],double *A,int m,int n) {

    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            out[i+j*m] =*(A+(i*n+j));
        }
    }
    return;
}
